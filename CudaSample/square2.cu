
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void square(float * d_in, float * d_out){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

int main(int arg, char** argv){

    const int ARRAY_SIZE = 8;
    const int ARRAY_BITES = ARRAY_SIZE * sizeof(float);

    float h_in[ARRAY_SIZE];
    float h_out[ARRAY_SIZE];

    for(int i=0;i<ARRAY_SIZE;i++){
        h_in[i] = float(i);
    }

    float * d_in;
    float * d_out;

    hipMalloc((void**)&d_in, ARRAY_BITES);
    hipMalloc((void**)&d_out, ARRAY_BITES);

    hipMemcpy(d_in, h_in, ARRAY_BITES, hipMemcpyHostToDevice);
    square<<<1, ARRAY_SIZE>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, ARRAY_BITES, hipMemcpyDeviceToHost);

    for(int i=0;i<ARRAY_SIZE; i++){
        printf("%f", h_out[i]);
        printf(((i%4)!=3?"\t":"\n"));
    }

    hipFree(d_in);
    hipFree(d_out);

    return 0;

}
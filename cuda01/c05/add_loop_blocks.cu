#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c){
    int tid = threadIdx.x;
    if(tid < N){
        c[tid] = a[tid] + b[tid];
    }
}

/**
 *  cd /home/tonye/cuda-workspace/cuda-app/cuda01/c05
 *  nvcc -o add_loop_blocks add_loop_blocks.cu
 */
int main(void){
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    //在GPu上分配内存
    HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&dev_c, N  *sizeof(int)));

    //在cpu赋值
    for(int i=0;i<N;i++){
        a[i] = i;
        b[i] = i * i;
    }

    //从cpu复制GPU
    HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<1, N>>>(dev_a, dev_b, dev_c);

    //从gpu复制到cpu
    HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

    //显示结果
    for(int i=0;i<N;i++){
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    //释放GPU内存
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
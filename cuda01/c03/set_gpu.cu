#include "../common/book.h"

int main(void){
    hipDeviceProp_t prop;
    int dev;
    HANDLE_ERROR(hipGetDevice(&dev));
    printf("ID of current CUDA device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 1;
    prop.minjor = 3;
    HANDLE_ERROR(hipChooseDevice(&dev, &prop));
    printf("ID of CUDA device closet to revision 1.3: %d\n");
    HANDLE_ERROR(hipSetDevice(dev));
}
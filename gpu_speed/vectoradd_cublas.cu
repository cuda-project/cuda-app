
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#define DATATYPE float

/*
cd /home/tonye/cuda-workspace/cuda-app/gpu_speed
nvcc vectoradd_cublas.cu -o vectoradd_cublas -lcublas
./vectoradd_cublas
 */
int main(int argc, char ** argv){
    const long ARRAY_SIZE = 65536;
    const long ARRAY_BYTIES = sizeof(DATATYPE) * ARRAY_SIZE;

    DATATYPE a[ARRAY_SIZE], b[ARRAY_SIZE], c[ARRAY_SIZE];

    for(long i =0;i<ARRAY_SIZE;i++){
        a[i] = DATATYPE(i);
        b[i] = DATATYPE(i);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    DATATYPE * d_a, * d_b;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_a, ARRAY_BYTIES);
    hipMalloc((void**)&d_b, ARRAY_BYTIES);
    DATATYPE alpha = 1.0;
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), a, 1, d_a, 1);
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), b, 1, d_b, 1);

    hipEventRecord(start, 0);
    hipblasSaxpy(handle, ARRAY_SIZE, &alpha, d_a, 1, d_b, 1);
    hipEventRecord(stop, 0);

    hipblasGetVector(ARRAY_SIZE, sizeof(DATATYPE), d_b, 1, c, 1);



    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("time elapsed: %f\n", elapsedTime);

//    for(int i=0;i<ARRAY_SIZE;i++){
//        printf("%f", c[i]);
//        printf(((i%4)!=3)?"\t":"\n");
//    }

    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);
}
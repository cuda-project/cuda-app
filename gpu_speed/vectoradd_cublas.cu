#include <stdio.h>
#include <hipblas.h>
#define DATATYPE float

#include "../include/gputimer.h"

/*
cd /home/tonye/cuda-workspace/cuda-app/gpu_speed
nvcc vectoradd_cublas.cu -o vectoradd_cublas -lcublas
./vectoradd_cublas
 */
int main(int argc, char ** argv){
    const long ARRAY_SIZE = 65536;
    const long ARRAY_BYTIES = sizeof(DATATYPE) * ARRAY_SIZE;

    DATATYPE a[ARRAY_SIZE], b[ARRAY_SIZE], c[ARRAY_SIZE];

    for(long i =0;i<ARRAY_SIZE;i++){
        a[i] = DATATYPE(i);
        b[i] = DATATYPE(i);
    }


    DATATYPE * d_a, * d_b;

    GpuTimer timer;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_a, ARRAY_BYTIES);
    hipMalloc((void**)&d_b, ARRAY_BYTIES);
    DATATYPE alpha = 1.0;
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), a, 1, d_a, 1);
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), b, 1, d_b, 1);

    timer.Start();
    hipblasSaxpy(handle, ARRAY_SIZE, &alpha, d_a, 1, d_b, 1);
    timer.Stop();

    hipblasGetVector(ARRAY_SIZE, sizeof(DATATYPE), d_b, 1, c, 1);

    printf("time elapsed: %f\n", timer.Elapsed());

//    for(int i=0;i<ARRAY_SIZE;i++){
//        printf("%f", c[i]);
//        printf(((i%4)!=3)?"\t":"\n");
//    }

    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);
}
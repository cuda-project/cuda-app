
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>
#define DATATYPE float

/*
cd /home/tonye/cuda-workspace/cuda-app/gpu_speed
nvcc vectoradd_cublas.cu -o vectoradd_cublas -lcublas
./vectoradd_cublas
 */
int main(int argc, char ** argv){
    const int ARRAY_SIZE = 10;
    const int ARRAY_BYTIES = sizeof(DATATYPE) * ARRAY_SIZE;

    DATATYPE a[ARRAY_SIZE], b[ARRAY_SIZE], c[ARRAY_SIZE];

    for(int i =0;i<ARRAY_SIZE;i++){
        a[i] = DATATYPE(i);
        b[i] = DATATYPE(i);
    }\

    DATATYPE * d_a, * d_b;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_a, ARRAY_BYTIES);
    hipMalloc((void**)&d_b, ARRAY_BYTIES);
    DATATYPE alpha = 1.0;
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), a, 1, d_a, 1);
    hipblasSetVector(ARRAY_SIZE, sizeof(DATATYPE), b, 1, d_b, 1);
    hipblasSaxpy(handle, ARRAY_SIZE, &alpha, d_a, 1, d_b, 1);
    hipblasGetVector(ARRAY_SIZE, sizeof(DATATYPE), d_b, 1, c, 1);


    for(int i=0;i<ARRAY_SIZE;i++){
        printf("%f", c[i]);
        printf(((i%4)!=3)?"\t":"\n");
    }

    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipblasDestroy(handle);
}
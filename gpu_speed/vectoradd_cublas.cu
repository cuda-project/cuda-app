
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

/*
cd /home/tonye/cuda-workspace/cuda-app/gpu_speed
nvcc vectoradd_cublas.cu -o vectoradd_cublas -lcublas
./vectoradd_cublas
 */
int main(int argc, char ** argv){
    const int ARRAY_SIZE = 10;
    const int ARRAY_BYTIES = sizeof(float) * ARRAY_SIZE;

    float a[ARRAY_SIZE], b[ARRAY_SIZE], c[ARRAY_SIZE];

    for(int i =0;i<ARRAY_SIZE;i++){
        a[i] = float(i);
        b[i] = float(i);
        c[i] = float(i);
    }

    float * d_a, * d_b;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipMalloc((void**)&d_a, ARRAY_BYTIES);
    hipMalloc((void**)&d_b, ARRAY_BYTIES);
    float alpha = 1.0;
    hipblasSetVector(ARRAY_SIZE, ARRAY_BYTIES, a, 1, d_a, 1);
    hipblasSetVector(ARRAY_SIZE, ARRAY_BYTIES, b, 1, d_b, 1);
    hipblasSaxpy(handle, ARRAY_SIZE, &alpha, d_a, 1, d_b, 1);
    hipblasSetVector(ARRAY_SIZE, ARRAY_BYTIES, d_b, 1, c, 1);

    for(int i=0;i<ARRAY_SIZE;i++){
        printf("%f", c[i]);
        printf(((i%4)!=3)?"\t":"\n");
    }

    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
}